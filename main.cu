
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <queue>
#include <sstream>
#include <string>
#include <ctime>
#include <assert.h>

struct vertex {
    int start;
    int numAdj;
    vertex() { numAdj = 0; start = -1; }
};


__global__ void parallelBFS(vertex* V, int* E, bool* q, bool* visited, bool* qNotEmpty) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (q[id] == true && visited[id] == false) {
        q[id] = false;
        visited[id] = true;
        __syncthreads();
        int start = V[id].start;
        int length = V[id].numAdj;
        if (length == 0) return;
        for (int i = start; i < start + length; i++) {
            int adjacent = E[i];
            if (visited[adjacent] == false) {
                q[adjacent] = true;
                __syncthreads();
                *qNotEmpty = true;
            }
        }
    }
    return;
}

void fillQueue(vertex* V, int* E, int n, std::queue<int> &q, bool* visited) {
    visited[n] = true;
    int start = V[n].start;
    int length = V[n].numAdj;
    if (length == 0) return;
    for (int i = start; i < start + length; i++) {
        if(!visited[E[i]]){
            q.push(E[i]);
        }
    }
}

void runBFS(vertex* V, int* E, int vertices, int edges, bool* visited) {
    for (int i = 0; i < vertices; i++) {
        visited[i] = false;
    }

    std::queue<int> q;
    fillQueue(V, E, 1, q, visited);
    
    while(!q.empty()) {
        int vert = q.front();
        q.pop();
        if (!visited[vert]) {
            fillQueue(V, E, vert, q, visited);
        }
    }

    /*for (int i = 1; i < vertices; i++) {
        std::cout << i << " " << visited[i] << std::endl;
    }*/
    
}

int main(int argc, char* argv[]) {

    if (argc != 2) {
        std::cerr << "Incorrect Usage, please use ./main [filename] " << std::endl;
    }
    std::string filename = argv[1];
    std::ifstream file(filename);
    std::string firstLine;
    getline(file, firstLine);
    std::stringstream ss(firstLine);

    int vertices, edges, numThreads;
    ss >> numThreads;
    getline(file, firstLine);
    std::stringstream ss1(firstLine);
    ss1 >> vertices >> edges;
    vertices++; edges++;
    vertex* V = new vertex[vertices];
    int* E = new int[edges];
    E[0] = 0;

    int currentVertex = 1;
    int counter = 1;
    V[1].start = 1;
    for (int i = 0; i < edges-1; i++) {
        std::string line;
        getline(file, line);
        std::stringstream ss2(line);
        int to, from, weight;
        ss2 >> to >> from >> weight;
        if (from != currentVertex) {
            currentVertex = from;
            V[from].start = counter;
        }
        V[from].numAdj++;
        E[counter] = to;
        counter++;
    }
    bool* visited = new bool[vertices];
    clock_t begin = clock();
    runBFS(V, E, vertices, edges, visited);
    clock_t end = clock();
    double timeSec = (end - begin) / static_cast<double>( CLOCKS_PER_SEC );
    std::cout << "Sequential Execution Time: " << timeSec << std::endl;

    bool* qNotEmpty = new bool;
    *qNotEmpty = true;

    bool* q = new bool[vertices];
    bool* visitedParallel = new bool[vertices];
    for (int i = 0; i < vertices; i++) {
        q[i] = false;
        visitedParallel[i] = false;
    }
    q[1] = true;

    vertex* deviceVertex;
    int* deviceEdges;
    bool* deviceQueue;
    bool* deviceVisited;
    bool* deviceQNotEmpty;

    hipMalloc(&deviceVertex, sizeof(vertex) * vertices);
    hipMalloc(&deviceEdges, sizeof(int) * edges);
    hipMalloc(&deviceQueue, sizeof(bool) * vertices);
    hipMalloc(&deviceVisited, sizeof(bool) * vertices);
    hipMalloc(&deviceQNotEmpty, sizeof(bool));

    hipMemcpy(deviceVertex, V, sizeof(vertex) * vertices, hipMemcpyHostToDevice);
    hipMemcpy(deviceEdges, E, sizeof(int) * edges, hipMemcpyHostToDevice);
    hipMemcpy(deviceQueue, q, sizeof(bool) * vertices, hipMemcpyHostToDevice);
    hipMemcpy(deviceVisited, visitedParallel, sizeof(bool) * vertices, hipMemcpyHostToDevice);
    hipMemcpy(deviceQNotEmpty, qNotEmpty, sizeof(bool), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(numThreads, 1, 1);
    dim3 numBlocks(vertices / numThreads + 1, 1, 1);

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    while(*qNotEmpty) {
        *qNotEmpty = false;
        hipMemcpy(deviceQNotEmpty, qNotEmpty, sizeof(bool), hipMemcpyHostToDevice);
        parallelBFS <<<numBlocks, threadsPerBlock>>> (deviceVertex, deviceEdges, deviceQueue, deviceVisited, deviceQNotEmpty);
        hipMemcpy(qNotEmpty, deviceQNotEmpty, sizeof(bool), hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    time *= 0.001;
    printf("Kernel Execution Time:  %3.5f s \n", time);

    hipMemcpy(visitedParallel, deviceVisited, sizeof(bool) * vertices, hipMemcpyDeviceToHost);
    for (int i = 1; i < vertices; i++) {
        //printf("%i, %i, %i\n", i, visitedParallel[i], visited[i]);
        assert(visitedParallel[i] == visited[i]);
    }
    std::cout << "Speedup of: " << timeSec / time << std::endl;
    std::cout << "Output matches serial execution" << std::endl;

    return 0;

}
